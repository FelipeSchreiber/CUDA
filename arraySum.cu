
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__global__ void add(int*a, int*b, int*c)
{
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

__global__ void fatorialAdd(int *a, int *b, int *c)
{
	int i;
	int maxA = a[blockIdx.x];
	int maxB = b[blockIdx.x];
	int fatA,fatB;
	fatA = fatB = 1;
	for(i = 0;i<fatA;i++)
		fatA *= (maxA - i);
	for(i = 0;i<fatB;i++)
		fatB *= (maxB - i);
	c[blockIdx.x] = fatA + fatB;
}

__global__ void random_ints(int *a, int shift)
{
	a[blockIdx.x] = blockIdx.x + shift;
}

long getMicrotime(){
	struct timeval currentTime;
	gettimeofday(&currentTime, NULL);
	return currentTime.tv_sec * (int)1e6 + currentTime.tv_usec;
}

#define N 10000000
int main(void)
{
	int*a, *b, *c;// host copies of a, b, c
	int*d_a, *d_b, *d_c;// device copies of a, b, c
	int i;
	int size = N * sizeof(int);
	long start,end;
	// Alloc space for device copies of a, b, c
	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);
	//Alloc space for host copies of a, b, c and setup input values
	a = (int *)malloc(size);
	random_ints<<<N,1>>>(d_a,13);
	hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
	b = (int *)malloc(size); 
	random_ints<<<N,1>>>(d_b,2);
	hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
	c = (int *)malloc(size);
	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
	// Launch add() kernel on GPU with N blocks
	long mediaTempo = 0;
	for(i = 0; i<1000;i++)
	{
		start = getMicrotime();	
		//add<<<N,1>>>(d_a, d_b, d_c);// Copy result back to host
		fatorialAdd<<<N,1>>>(d_a, d_b, d_c);
		hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);// Cleanup
		end = getMicrotime();
		mediaTempo += (end - start); 
	}
	printf("\nTOTAL TIME: %ld\n",mediaTempo/1000);
	for(i = 1; i<4; i++)
		printf("\nSUM of %i + %i = %i\n",a[N-i],b[N-i],c[N-i]);
	
	int j,k;
	int maxA = 0;
	int maxB = 0;
	int fatA,fatB;
	mediaTempo = 0;
	for(k = 0; k<1000; k++)	
	{
		start = getMicrotime();
		for(i = 0; i < N; i++)
		{
			fatA = fatB = 1;
			maxA = a[i];
			maxB = b[i];
			for(j = 0;j<fatA;j++)
				fatA *= (maxA - j);
			for(j = 0;j<fatB;j++)
				fatB *= (maxB - j);
			c[i] = fatA + fatB;
		}
		end = getMicrotime();
		mediaTempo += (end - start); 
	}
	printf("\nTOTAL TIME: %ld\n",mediaTempo/1000);
	for(i = 1; i<4; i++)
		printf("\nSUM of %i + %i = %i\n",a[N-i],b[N-i],c[N-i]);
	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}

